#include "hip/hip_runtime.h"
#include "KernelPruning.h"

#include "cub.cuh"
#include "utils.h"

#include <thrust/device_vector.h>
#include <vector>

#include <cstdio>
#include "gputimer.h"

#define CAST(dv) (thrust::raw_pointer_cast(dv.data()))

debugger dbg;


/**
 * @brief [This is a device method which returns a boolean value depending on the whether the condition is satisfied]
 * @details [In this method, the condition is whether the degree is greater than the threshold. If yes, it returns true else false.]
 * 
 * @param degree [degree of the current node]
 * @param degreeThreshold [threshold condition]
 * 
 * @return [nothing is returned]
 */
__device__
bool condition(unsigned degree,unsigned degreeThreshold)
{
	return (degree > degreeThreshold);
}

/**
 * @brief [This Kernel is used to filter the vertices having degree > degreeThreshold]
 * @details [We first check the degree of the row vertices , if the result is 0, then set the result in the current position,
 * 			else we check the column vertices and then set the corresponding result.]
 * 
 * @param d_rows [This array contains the "source" vertices.]
 * @param d_cols [This array contains the "destination" vertices] 
 * @param d_degree [This array contains the degree of each of the vertex]
 * @param d_filterArray [This array is used to output the result i.e 0 if condition fails else 1.]
 * @param degreeThreshold [Condition threshold]
 * @param count [Count of number of columns.]
 */
__global__ 
void KernelFilterColOffset(unsigned *d_rows,unsigned *d_cols,unsigned *d_degree,unsigned *d_filterArray,unsigned degreeThreshold,int count)
{
	//thread Id of the current thread;
	int tid = threadIdx.x + blockIdx.x*blockDim.x;

	//bounds check
	if(tid >= count)
		return;
	
	//obtain the row first
	unsigned currentRowValue = d_rows[tid];

	//degree corresponding to the row.
	unsigned degree_RowValue = d_degree[currentRowValue];


	int result = condition(degree_RowValue , degreeThreshold);

	if(result == 0)
	{
		d_filterArray[tid] = result;
	}
	else
	{

		//obtain the column value
		unsigned currentColValue = d_cols[tid];

		//obtain the degree corresponding to the column.
		unsigned degree_ColValue = d_degree[currentColValue];

		//Evaluate the condition and obtain the results
		int result = condition(degree_ColValue , degreeThreshold);

		//set the filter value.
		d_filterArray[tid] = result;
	}

}

/**
 * @brief [This Kernel involved each thread updating the degree of the remaining unfiltered Vertices]
 * @details [We update the d_degree[d_uniqueOut[tid]] = d_countsOut[tid]]
 * 
 * @param d_degree [original degree array]
 * @param d_uniqueOut [unique rows offset Array]
 * @param d_countsOut [unique Counts array corresponding to each RowOffset]
 * @param count_elements [Number of element, used for boundary condition]
 * @return [description]
 */
__global__
void KernelUpdateDegree(unsigned *d_degree,unsigned *d_uniqueOut,unsigned *d_countsOut,int count_elements)
{
		int tid = threadIdx.x + blockDim.x*blockIdx.x;
		if(tid >= count_elements)
				return;
		d_degree[d_uniqueOut[tid]] = d_countsOut[tid];
}


void DeviceReset()
{
	CudaError(hipDeviceReset());
}
/**
 * @brief [This method is used to invoke the hipcub::select method]
 * @details [We first determine the buffer size in the first invocation. We then pass either NULL or the buffer array as argument]
 * 
 * @param d_temp_storage1 [contains NULL]
 * @param d_temp_storage2 [buffer array]
 * @param d_in [Input array]
 * @param d_flags [Input flags array]
 * @param d_out [Output array]
 * @param d_num_selected_out [number of values selected]
 * @param num_items [number of items]
 */
static void SelectMethodHelper(void *d_temp_storage1,unsigned *d_temp_storage2,unsigned *d_in,unsigned *d_flags,unsigned *d_out,int *d_num_selected_out,int num_items)
{
		size_t buffer_size = 0;

		unsigned temp;
		//Ist invocation to determine the buffer size required.
		CudaError(hipcub::DeviceSelect::Flagged(d_temp_storage1,buffer_size,d_in,d_flags,d_out,d_num_selected_out,num_items));

		void *buffer =  d_temp_storage2;

		//Actual invocation of the flag array.
		CudaError(hipcub::DeviceSelect::Flagged(buffer,buffer_size,d_in,d_flags,d_out,d_num_selected_out,num_items));

		CudaError(hipDeviceSynchronize());

}

/**
 * @brief [This method is used to invoke the hipcub::RunLengthEncode method]
 * @details [We first determine the buffer size in the first invocation. We then pass either NULL or the buffer array as argument]
 * 
 * @param d_temp_storage1 [contains NULL]
 * @param d_temp_storage2 [contains Buffer Array]
 * @param d_in [Input Array]
 * @param d_uniqueOut [Output_array]
 * @param d_countsOut [Counts for each element]
 * @param d_num_selected_out [number of unique element]
 * @param num_items [number of input items]
 */
static void RunLengthEncodingHelper(void *d_temp_storage1,unsigned *d_temp_storage2,unsigned *d_in,unsigned *d_uniqueOut,unsigned *d_countsOut,int *d_num_runs_out,int num_items)
{
		size_t buffer_size = 0;

		unsigned temp;

		//Ist invocation of the method to determine the space required
		CudaError(hipcub::DeviceRunLengthEncode::Encode(d_temp_storage1,buffer_size,d_in,d_uniqueOut,d_countsOut,d_num_runs_out,num_items));

		void *buffer = d_temp_storage2;  //determing the buffer 

		//debug("buffer_size ", buffer_size);

		CudaError(hipcub::DeviceRunLengthEncode::Encode(buffer,buffer_size,d_in,d_uniqueOut,d_countsOut,d_num_runs_out,num_items));

}

/**
 * @brief [brief description]
 * @details [long description]
 * 
 * @param graph [description]
 * @param degreeThreshold [description]
 */
float KernelPruningWrapper(csr_graph &graph,int degreeThreshold)
{
		float totalTime = 0;

		GpuTimer gpuTimer;

		#define BLOCKSIZE 128    //BlockSize 
		#define GRIDSIZE(n) (ceil((double)n/BLOCKSIZE)) //GridSize 
		#define max(a,b) ((a < b)? b : a)
		//Declare the variables to hold the corresponding sizes of the auxilliary array.

		size_t aux_filter_rows;
		size_t aux_filter_run_length;

		void *d_temp_storage = NULL;

		bool toBePruned = true;

		bool flag_type; //flagtype is either 0 or 1. 0 indicates no vertices left, 1 indicates all remaining vertices are left

		#ifdef INFO
			int num_vertices_pruned = 0;
		#endif

		//Get the dimensions of the elements
		int count_rowoffset = graph.degree->size();
		int count_columns   = graph.rows->size();


		//Declare the corresponding containers for the thrust vectors.
		thrust::device_vector<unsigned> d_rows(graph.rows->begin(),graph.rows->end());
		thrust::device_vector<unsigned> d_cols(graph.columns->begin(),graph.columns->end());
		thrust::device_vector<unsigned> d_degree(graph.degree->begin(),graph.degree->end());

		thrust::device_vector<unsigned> d_filter(max(count_rowoffset,count_columns));
		thrust::device_vector<unsigned> d_uniqueOut(max(count_rowoffset,count_columns));
		thrust::device_vector<unsigned> d_countsOut(max(count_rowoffset,count_columns));

		thrust::device_vector<int> d_num_selected_out(1);

		//Get tempBufferSize for d_rowOffset
		CudaError(hipcub::DeviceSelect::Flagged(d_temp_storage,aux_filter_rows,CAST(d_cols),CAST(d_filter),CAST(d_cols),CAST(d_num_selected_out),count_columns));
		CudaError(hipcub::DeviceRunLengthEncode::Encode(d_temp_storage,aux_filter_run_length,CAST(d_rows),CAST(d_uniqueOut),CAST(d_countsOut),CAST(d_num_selected_out),count_columns));

		//Allocate buffer size
		size_t allocate_temporary_buffer_size = max(aux_filter_rows,aux_filter_run_length);

		//This device_vector is used as a buffer in the cub runtime calls.
		thrust::device_vector<unsigned> d_temp_buffer(allocate_temporary_buffer_size/4);

		//debug("max_size_allocated =",allocate_temporary_buffer_size , "bytes");
		//debug("Unique Vertexes count:", count_rowoffset);
		//debug("Columns count :" , count_columns);

		if(count_rowoffset == 0 && count_columns ==0)
			toBePruned = false;

		gpuTimer.Start();

		//Run until this flag is false.
		while(toBePruned)
		{

			//Step 1: Apply  the filtering Kernel to store valyues 1 and 0 in the d_filter array corresponding to the columns
			KernelFilterColOffset<<<GRIDSIZE(count_columns),BLOCKSIZE>>>(CAST(d_rows),CAST(d_cols),CAST(d_degree),CAST(d_filter),degreeThreshold,count_columns);

			CudaError(hipDeviceSynchronize());

			//Step 2: Use the filter array to Select the Columns 
			SelectMethodHelper(d_temp_storage,CAST(d_temp_buffer),CAST(d_cols),CAST(d_filter),CAST(d_cols),CAST(d_num_selected_out), count_columns);

			//Step 3: Use the previous filter array to Select the rows too.
			SelectMethodHelper(d_temp_storage,CAST(d_temp_buffer),CAST(d_rows),CAST(d_filter),CAST(d_rows),CAST(d_num_selected_out), count_columns);


			//get the new count of the rows
			int new_columns_count = d_num_selected_out[0];

			if((new_columns_count == 0) || (new_columns_count == count_columns))
			{
				//debug("Finished : Final Columns Count",count_columns , " Final Unique Rows count = ", count_rowoffset);

				if(new_columns_count == 0)
				{
					flag_type = 0;
					//debug("All vertices are removed!!!");
				}
				else
				{
					flag_type = 1;
					//debug("Some vertices remain.")
				}

				break;
			}

			//Step 4: Do a runLengthEncoding on the Rows Array.
			RunLengthEncodingHelper(d_temp_storage,CAST(d_temp_buffer),CAST(d_rows),CAST(d_uniqueOut),CAST(d_countsOut),CAST(d_num_selected_out),new_columns_count);

			count_rowoffset = d_num_selected_out[0]; //update the count of rowoffset


			//Step 5: Update the degree of the degree Array based on the d_uniqueOut and d_countsOut
			KernelUpdateDegree<<<GRIDSIZE(new_columns_count),BLOCKSIZE>>>(CAST(d_degree),CAST(d_uniqueOut),CAST(d_countsOut),count_rowoffset);

			CudaError(hipDeviceSynchronize());

			count_columns = new_columns_count;

		}

		totalTime += gpuTimer.StopGetTime();


		//Do something with the values here

		if(flag_type == 1)
		{
			//Make host_vectors from device vectors
			thrust::host_vector<unsigned> prunedRows(d_rows.begin(),d_rows.begin() + count_columns);
			thrust::host_vector<unsigned> prunedCols(d_cols.begin(),d_cols.begin() + count_columns);
			thrust::host_vector<unsigned> prunedDegree(d_degree.begin(),d_degree.end());

			//Clear the current host graph configuration
			graph.rows->clear();
			graph.columns->clear();
			graph.degree->clear();

			//Fill the pruned graph into the graph.(This will update the graph with the pruned version from the GPU)
			graph.rows->insert(graph.rows->end(),prunedRows.begin(),prunedRows.end());
			graph.columns->insert(graph.columns->end(),prunedCols.begin(),prunedCols.end());
			graph.degree->insert(graph.degree->end(),prunedDegree.begin(),prunedDegree.end());

			
			//Free the arrays

			prunedRows.clear();
			prunedCols.clear();
			prunedDegree.clear();

		}
		else
		{
			graph.rows->clear();
			graph.columns->clear();
			graph.degree->clear();
		}



		d_rows.clear();
		d_cols.clear();
		d_degree.clear();
		d_filter.clear();
		d_uniqueOut.clear();
		d_countsOut.clear();
		d_num_selected_out.clear();
		d_temp_buffer.clear();

		return totalTime;
		

}
